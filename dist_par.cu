#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <errno.h>
#include <sys/time.h>

char *aloca_sequencia(int n)
{
	char *seq;

	seq = (char *) malloc((n + 1) * sizeof(char));
	if (seq == NULL)
	{
		perror("\nErro na alocação de estruturas\n") ;
		exit(1) ;
	}
	return seq;
}

int *aloca_matriz(int totalMatriz)
{
	int *mat;

	mat = (int *) malloc((totalMatriz) * sizeof(int));
	if (mat == NULL)
	{
		perror("\nErro na alocação de estruturas\n") ;
		exit(1) ;
	}
	return mat;
}

void distancia_edicao(int n, int m, char *s, char *r, int *d)
{
	int nADiag,			// Número de anti-diagonais
		tamMaxADiag,	// Tamanho máximo (número máximo de células) da anti-diagonal
		aD,				// Anti-diagonais numeradas de 2 a nADiag + 1
		k, i, j,
		t, a, b, c, min;

	nADiag = n + m - 1;
	tamMaxADiag = n;

	// Para cada anti-diagonal
	for (aD = 2; aD <= nADiag + 1; aD++)
	{
		// Para cada célula da anti-diagonal aD
		for (k = 0; k < tamMaxADiag; k++)
		{
			// Calcula índices i e j da célula (linha e coluna)
			i = n - k;
			j = aD - i;
			
			// Se é uma célula válida
			if (j > 0 && j <= m)
			{
				t = (s[i] != r[j] ? 1 : 0);
				a = d[i][j-1] + 1;
				b = d[i-1][j] + 1;
				c = d[i-1][j-1] + t;
				// Calcula d[i][j] = min(a, b, c)
				if (a < b)
					min = a;
				else
					min = b;
				if (c < min)
					min = c;
				d[i][j] = min;
			}
		}
	}
}

void libera(int n, char *s, char *r, int *d)
{
	int i;

	free(s);
	free(r);
	free(d);
}

__global__ void diagonal_Par(int *d, int *tamanho){
	int idGlobal = blockIdx.x * blockDim.x + threadIdx.x;


	return;
}

int main(int argc, char **argv)
{
	int  n,		// Tamanho da sequência s
		 m,		// Tamanho da sequência r
		 *d,	// Matriz de distâncias com tamanho (n+1)*(m+1)
		 i, j;
	char *s,	// Sequência s de entrada (vetor com tamanho n+1)
		 *r;	// Sequência r de entrada (vetor com tamanho m+1)
	FILE *arqEntrada ;	// Arquivo texto de entrada

	int *d_m;

	if(argc != 2)
	{
		printf("O programa foi executado com argumentos incorretos.\n") ;
		printf("Uso: ./dist_seq <nome arquivo entrada>\n") ;
		exit(1) ;
	}

	// Abre arquivo de entrada
	arqEntrada = fopen(argv[1], "rt") ;

	if (arqEntrada == NULL)
	{
		printf("\nArquivo texto de entrada não encontrado\n") ;
		exit(1) ;
	}

	// Lê tamanho das sequências s e r
	fscanf(arqEntrada, "%d %d", &n, &m) ;

	// Aloca vetores s e r
	s = aloca_sequencia(n);
	r = aloca_sequencia(m);
	// Aloca matriz d
	d = aloca_matriz((n+1)*(m+1));

	// Lê sequências do arquivo de entrada
	s[0] = ' ';
	r[0] = ' ';
	fscanf(arqEntrada, "%s", &(s[1])) ;
	fscanf(arqEntrada, "%s", &(r[1])) ;
	
	// Fecha arquivo de entrada
	fclose(arqEntrada) ;

	struct timeval h_ini, h_fim;
	gettimeofday(&h_ini, 0);

	// Inicializa matriz de distâncias d
	for (i = 0; i <= n; i++)
	{
        d[i] = i;
    }
    
    for (j = 1; j <= m; j++)
	{
		d[(m*j)+j] = j;
	}

	hipMalloc((void **) &d_m, sizeof(int) * n*m);
	hipMemcpy(d_m, d, sizeof(d), hipMemcpyHostToDevice);
	int *max;
	*max = sizeof(int) *n*m;

	// numero de blocos == sequencia S 
	diagonal_Par <<<n, 512 >>>(d_m, max);
	

	// Calcula distância de edição entre sequências s e r, por anti-diagonais
	distancia_edicao(n, m, s, r, d);

	gettimeofday(&h_fim, 0);
	// Tempo de execução na CPU em milissegundos
	long segundos = h_fim.tv_sec - h_ini.tv_sec;
	long microsegundos = h_fim.tv_usec - h_ini.tv_usec;
	double tempo = (segundos * 1e3) + (microsegundos * 1e-3);

	printf("Distância=%d\n", d[n][m]);
	printf("Tempo CPU = %.2fms\n", tempo);

	// Libera vetores s e r e matriz d
	libera(n, s, r, d);

	return 0;
}